
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 3.0f;
  }

  // Run kernel on 1M elements on the GPU
  auto start=high_resolution_clock::now();
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  auto stop=high_resolution_clock::now();
  auto duration=duration_cast<microseconds>(stop-start);

  cout<<"duration:"<<duration.count()<<endl;

  // Check for errors (all values should be 3.0f)
  for(int i=0; i<10; i++)
  {
    cout<<y[i]<<" ";
  }cout<<endl;


  // float maxError = 0.0f;
  // for (int i = 0; i < N; i++)
    // maxError = fmax(maxError, fabs(y[i]-3.0f));
  // std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}
