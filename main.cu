
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;

  // y[index]=1.0;
  // x[index]=stride;

  for (int i = index; i < n; i+=stride)
    y[i] = x[i] + y[i];
}

void cpu_add(int n,float *x, float*y){

  for(int i=0; i<n; i++) {
    y[i]=x[i]+y[i];
  }
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 2.0f;
    y[i] = 1.0f;
  }

  // on CPU
  float* x_c = new float[N];
  float* y_c = new float[N];
  for(int i=0; i<N; i++)
  {
    x_c[i]=2.0f;
    y_c[i]=1.0f;
  }

  auto start=high_resolution_clock::now();
  cpu_add(N,x_c,y_c);
  auto stop=high_resolution_clock::now();
  auto duration=duration_cast<milliseconds>(stop-start);

  cout<<"CPU duration:"<<duration.count()<<endl;
  cout<<"y_c:"<<endl;
  for(int i=0; i<10; i++) {
    cout<<y_c[i]<<" ";
  }cout<<endl;
  cout<<"x_c:"<<endl;
  for(int i=0; i<10; i++) {
    cout<<x_c[i]<<" ";
  }cout<<endl;


  // Run kernel on 1M elements on the GPU
  start=high_resolution_clock::now();

  int blockSize=256;
  int numBlocks=(N+blockSize-1)/blockSize;
  cout<<"numBlocks"<<numBlocks<<endl;
  add<<<numBlocks, blockSize>>>(N, x, y);
  // add<<<1, 256>>>(N, x, y);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  stop=high_resolution_clock::now();
  duration=duration_cast<milliseconds>(stop-start);

  cout<<"GPU duration:"<<duration.count()<<endl;
  // Check for errors (all values should be 3.0f)
  cout<<"y:"<<endl;
  for(int i=0; i<10; i++) {
    cout<<y[i]<<" ";
  }cout<<endl;
  cout<<"x:"<<endl;
  for(int i=0; i<10; i++) {
    cout<<x[i]<<" ";
  }cout<<endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  delete [] x_c;
  delete [] y_c;
  
  return 0;
}
