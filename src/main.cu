
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
struct array2d{
  int width;
  int height;
  float* data;
  array2d(int width,int height):width(width),height(height){
    hipMallocManaged(&data,width*height*sizeof(float));
  }
  ~array2d(){
    hipFree(&data);
  }

};
struct array3d{
  int size_0;
  int size_1;
  int size_2;
  int length;
  float* data;
  array3d(int size_0,int size_1, int size_2)
    :size_0(size_0),size_1(size_1),size_2(size_2)
  {
    length=size_0*size_1*size_2;
    cout<<"calling hipMallocManaged"<<endl;
    hipMallocManaged(&data,length*sizeof(float));
  }
  ~array3d(){
    hipFree(&data);
  }
  void show(){
    for(int _i0=0; _i0 < size_0; _i0++){
      // printing slice
      for(int _i1=0; _i1 < size_1; _i1++){
        for(int _i2=0; _i2 < size_2; _i2++){
          cout<<data[_i0*size_1*size_2 + _i1*size_2 + _i2]<<" ";
        }cout<<endl;
      }cout<<"------"<<endl;
    }
  }

};
__device__
float GetElement(const array3d &arr, int i_0,int i_1,int i_2)
{
  return arr.data[i_0*arr.size_1*arr.size_2 + i_1*arr.size_2 + i_2];
}
__device__ void SetElement(array3d &arr, int i_0, int i_1, int i_2, float value)
{
  arr.data[i_0*arr.size_1*arr.size_2 + i_1*arr.size_2 + i_2]=value;
}

__global__
void add(array3d arr1, array3d arr2)
{
  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < arr1.length; i+=stride){

    // unravel index
    int _i_ur_0=i/(arr1.size_1*arr1.size_2);
    int _i_ur_1=(i-(arr1.size_1*arr1.size_2*_i_ur_0))/(arr1.size_2);
    int _i_ur_2=i%arr1.size_2;

    float e=GetElement(arr1,_i_ur_0,_i_ur_1,_i_ur_2);

    if(_i_ur_1+1<arr2.size_1)
      SetElement(arr2,_i_ur_0,_i_ur_1+1,_i_ur_2, e);

  }
}
int main(void)
{

  int N = 3;
  array3d arr1(N,N,10);
  array3d arr2(N,N,10);

  // initialize x and y arrays on the host
  int val=0;
  for (int i = 0; i < arr1.length; i++) {
    arr1.data[i] = val++;
    arr2.data[i] = 0.0f;
  }
  // Run kernel on 1M elements on the GPU
  int blockSize=256;
  int numBlocks=(N*N+blockSize-1)/blockSize;
  cout << "numBlocks => " << numBlocks << endl;
  add<<<numBlocks, blockSize>>>(arr1, arr2);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  cout<<"arr1:"<<endl;
  arr1.show();
  cout<<"arr2:"<<endl;
  arr2.show();
  return 0;
}
