
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
typedef struct{
  int width;
  int height;
  float* data;
} array2d;

__global__
void add(array2d arr1, array2d arr2)
// void add(int n,float*x, float*y)
{
  // int row=blockIdx.y*blockDim.y+threadIdx.y;
  // int col=blockIdx.x*blockDim.x+threadIdx.x;
  // arr1.data[arr1.width*row+col]+=arr2.data[arr2.width*row+col];

  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < arr1.width; i+=stride){
    arr1.data[i] = arr1.data[i]+arr2.data[i];
  }

}
int main(void)
{
  int N = 10;
  array2d arr1;
  arr1.width=N;
  arr1.height=N;

  array2d arr2;
  arr2.width=N;
  arr2.height=N;

  hipMallocManaged(&arr1.data,N*N*sizeof(float));
  hipMallocManaged(&arr2.data,N*N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N*N; i++) {
    arr1.data[i] = 3.0f;
    arr2.data[i] = 3.0f;
  }
  // Run kernel on 1M elements on the GPU
  int blockSize=256;
  int numBlocks=(N+blockSize-1)/blockSize;
  cout << "numBlocks => " << numBlocks << endl;
  // dim3 dimBlock(blockSize,blockSize);
  // dim3 dimGrid(arr1.width/dimBlock.x,arr1.height/dimBlock.y);
  // add<<<dimGrid, dimBlock>>>(arr1.data, arr2.data);
  add<<<numBlocks, blockSize>>>(arr1, arr2);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  cout<<"arr1:"<<endl;
  // arr1.print();
  for(int i=0; i < N; i++){
    for(int j=0; j < N; j++){
      cout<<arr1.data[i*arr1.width+j]<<"  ";
    }cout<<endl;
  }
  // cout<<"arr2:"<<endl;
  // arr2.print();
  return 0;
}
