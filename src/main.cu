
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
struct array2d{
  int width;
  int height;
  float* data;
  array2d(int width,int height):width(width),height(height){
    hipMallocManaged(&data,width*height*sizeof(float));
  }
  ~array2d(){
    hipFree(&data);
  }

};

__global__
void add(array2d arr1, array2d arr2)
// void add(int n,float*x, float*y)
{
  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < arr1.width*arr1.height; i+=stride){
    // arr1.data[i] = arr1.data[i]+arr2.data[i];
    // arr1.data[i] = blockIdx.x;

    // unravel index
    int row=i/arr1.width;
    int col=i%arr1.width;

    // arr1.data[row*arr1.width+col]=threadIdx.x;
    arr1.data[row*arr1.width+col]+=arr2.data[row*arr1.width+col];
  }
}
int main(void)
{
  int N = 10;
  array2d arr1(N,N);
  array2d arr2(N,N);

  // initialize x and y arrays on the host
  for (int i = 0; i < N*N; i++) {
    arr1.data[i] = 3.0f;
    arr2.data[i] = 5.0f;
  }
  // Run kernel on 1M elements on the GPU
  int blockSize=256;
  int numBlocks=(N*N+blockSize-1)/blockSize;
  cout << "numBlocks => " << numBlocks << endl;
  add<<<numBlocks, blockSize>>>(arr1, arr2);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  cout<<"arr1:"<<endl;
  // arr1.print();
  for(int i=0; i < N; i++){
    for(int j=0; j < N; j++){
      cout<<arr1.data[i*arr1.width+j]<<"  ";
    }cout<<endl;
  }
  // cout<<"arr2:"<<endl;
  // arr2.print();
  return 0;
}
