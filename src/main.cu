
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
struct array2d{
  float* data;
  int width;
  int height;
  array2d(int width,int height):width(width),height(height)
  {
    hipMallocManaged(&data,width*height*sizeof(float));
  };
  void print(){
    for(int i=0; i < width; i++)
    {
      for(int j=0; j < height; j++){
        cout<<data[width*i+j]<<"  ";
      }cout<<endl;
    }
  }
  ~array2d(){
    hipFree(data);
  }
};
__global__
// void add(array2d &arr1, array2d &arr2)
void add(int n,float*x, float*y)
{
  // int row=blockIdx.y*blockDim.y+threadIdx.y;
  // int col=blockIdx.x*blockDim.x+threadIdx.x;
  // arr1.data[arr1.width*row+col]+=arr2.data[arr2.width*row+col];

  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < n; i+=stride){
    x[i] = x[i]+y[i];
  }

}
int main(void)
{
  int N = 10;
  array2d arr1(N,N);
  array2d arr2(N,N);

  // initialize x and y arrays on the host
  for (int i = 0; i < N*N; i++) {
    arr1.data[i] = 3.0f;
    arr2.data[i] = 3.0f;
  }
  // Run kernel on 1M elements on the GPU
  int blockSize=256;
  int numBlocks=(N+blockSize-1)/blockSize;
  cout << "numBlocks => " << numBlocks << endl;
  // dim3 dimBlock(blockSize,blockSize);
  // dim3 dimGrid(arr1.width/dimBlock.x,arr1.height/dimBlock.y);
  // add<<<dimGrid, dimBlock>>>(arr1.data, arr2.data);
  add<<<numBlocks, blockSize>>>(N,arr1.data, arr2.data);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  cout<<"arr1:"<<endl;
  arr1.print();
  // cout<<"arr2:"<<endl;
  // arr2.print();
  return 0;
}
