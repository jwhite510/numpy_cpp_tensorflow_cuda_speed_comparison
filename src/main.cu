
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
struct array2d{
  int width;
  int height;
  float* data;
  array2d(int width,int height):width(width),height(height){
    hipMallocManaged(&data,width*height*sizeof(float));
  }
  ~array2d(){
    hipFree(&data);
  }

};
struct array3d{
  int size_0;
  int size_1;
  int size_2;
  int length;
  float* h_data;
  float* d_data;
  array3d(int size_0,int size_1, int size_2)
    :size_0(size_0),size_1(size_1),size_2(size_2)
  {
    length=size_0*size_1*size_2;
    // allocate memory on device
    hipMalloc(&d_data,length*sizeof(float));
    // allocate memory on host
    h_data = new float[length];
    // cudaMallocManaged(&data,length*sizeof(float));
  }
  void CopyToHost(){
    hipMemcpy(h_data,d_data,length*sizeof(float),hipMemcpyDeviceToHost);
  }
  void CopyToDevice(){
    hipMemcpy(d_data,h_data,length*sizeof(float),hipMemcpyHostToDevice);
  }
  ~array3d(){
    delete [] h_data;
    hipFree(d_data);
  }
  void show(){
    for(int _i0=0; _i0 < size_0; _i0++){
      // printing slice
      for(int _i1=0; _i1 < size_1; _i1++){
        for(int _i2=0; _i2 < size_2; _i2++){
          cout<<h_data[_i0*size_1*size_2 + _i1*size_2 + _i2]<<" ";
        }cout<<endl;
      }cout<<"------"<<endl;
    }
  }

};
__device__
float GetElement(const array3d &arr, int i_0,int i_1,int i_2)
{
  return arr.d_data[i_0*arr.size_1*arr.size_2 + i_1*arr.size_2 + i_2];
}
__device__ void SetElement(array3d &arr, int i_0, int i_1, int i_2, float value)
{
  arr.d_data[i_0*arr.size_1*arr.size_2 + i_1*arr.size_2 + i_2]=value;
}

__global__
void add(int N,float* d_array)
{
  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < N; i+=stride){
    d_array[i]=2*d_array[i];
  }
}
int main(void)
{

  int N=100;
  float* h_array= new float[N];
  float* d_array;
  hipMalloc(&d_array,N*sizeof(float));

  int val=0;
  for(int i=0; i < N; i++){
    h_array[i]=val++;
  }

  // HOST TO DEVICE
  hipMemcpy(d_array,h_array,N*sizeof(float),hipMemcpyHostToDevice);

  int blockSize=256;
  int numBlocks=(N+blockSize-1)/blockSize;
  add<<<numBlocks, blockSize>>>(N,d_array);

  // DEVICE TO HOST
  hipMemcpy(h_array,d_array,N*sizeof(float),hipMemcpyDeviceToHost);

  for(int i=0; i < 10; i++){
    cout<<h_array[i]<<" ";
  }cout<<endl;

  return 0;
}
