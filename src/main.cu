
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
struct array2d{
  int width;
  int height;
  float* data;
  array2d(int width,int height):width(width),height(height){
    hipMallocManaged(&data,width*height*sizeof(float));
  }
  ~array2d(){
    hipFree(&data);
  }

};
struct array3d{
  int size_0;
  int size_1;
  int size_2;
  int length;
  float* h_data;
  float* d_data;
  void CopyToHost(){
    hipMemcpy(h_data,d_data,length*sizeof(float),hipMemcpyDeviceToHost);
  }
  void CopyToDevice(){
    hipMemcpy(d_data,h_data,length*sizeof(float),hipMemcpyHostToDevice);
  }
  void show(){
    for(int _i0=0; _i0 < size_0; _i0++){
      // printing slice
      for(int _i1=0; _i1 < size_1; _i1++){
        for(int _i2=0; _i2 < size_2; _i2++){
          cout<<h_data[_i0*size_1*size_2 + _i1*size_2 + _i2]<<" ";
        }cout<<endl;
      }cout<<"------"<<endl;
    }
  }

};
void construct(array3d &arr, int size_0,int size_1, int size_2)
{
  arr.size_0=size_0;
  arr.size_1=size_1;
  arr.size_2=size_2;
  arr.length=size_0*size_1*size_2;
  // allocate memory on device
  hipMalloc(&arr.d_data,arr.length*sizeof(float));
  // allocate memory on host
  arr.h_data = new float[arr.length];
}
void ToDevice(array3d &arr){
  hipMemcpy(arr.d_data,arr.h_data,arr.length*sizeof(float),hipMemcpyHostToDevice);
}
void ToHost(array3d &arr){
  hipMemcpy(arr.h_data,arr.d_data,arr.length*sizeof(float),hipMemcpyDeviceToHost);
}
void destruct(array3d &arr){
  delete [] arr.h_data;
  hipFree(arr.d_data);
}
__device__
float GetElement(const array3d arr, int i_0,int i_1,int i_2)
{
  return arr.d_data[i_0*arr.size_1*arr.size_2 + i_1*arr.size_2 + i_2];
}
__device__ void SetElement(array3d arr, int i_0, int i_1, int i_2, float value)
{
  arr.d_data[i_0*arr.size_1*arr.size_2 + i_1*arr.size_2 + i_2]=value;
}

__global__
void add(array3d arr1, array3d arr2)
{
  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < arr1.length; i+=stride){

    // unravel index
    int _i_ur_0=i/(arr1.size_1*arr1.size_2);
    int _i_ur_1=(i-(arr1.size_1*arr1.size_2*_i_ur_0))/(arr1.size_2);
    int _i_ur_2=i%arr1.size_2;

    float e=GetElement(arr1,_i_ur_0,_i_ur_1,_i_ur_2);
    if(_i_ur_1+1<arr2.size_1)
      SetElement(arr2,_i_ur_0,_i_ur_1+1,_i_ur_2, e);
    // SetElement(arr2,_i_ur_0,_i_ur_1,_i_ur_2, i);

  }
}

struct FDTD{
  array3d arr1;
  array3d arr2;
  FDTD(){
    int N = 3;
    construct(arr1, N,N,10);
    construct(arr2, N,N,10);

    // initialize x and y arrays on the host
    int val=0;
    for (int i = 0; i < arr1.length; i++) {
      arr1.h_data[i] = val++;
      arr2.h_data[i] = 0.0f;
    }
  }
  void run(){
    arr1.CopyToDevice();
    arr2.CopyToDevice();

    int blockSize=256;
    int numBlocks=(arr1.length+blockSize-1)/blockSize;
    cout << "numBlocks => " << numBlocks << endl;
    add<<<numBlocks, blockSize>>>(arr1, arr2);
    arr1.CopyToHost();
    arr2.CopyToHost();

    // Wait for GPU to finish before accessing on host
    cout<<"arr1:"<<endl;
    arr1.show();
    cout<<"arr2:"<<endl;
    arr2.show();
  }
  ~FDTD(){
    destruct(arr1);
    destruct(arr2);
  }
};
int main(void)
{
  FDTD fdtd;
  fdtd.run();

  return 0;
}
