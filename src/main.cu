
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
struct array2d{
  int width;
  int height;
  float* data;
  array2d(int width,int height):width(width),height(height){
    hipMallocManaged(&data,width*height*sizeof(float));
  }
  ~array2d(){
    hipFree(&data);
  }

};
struct array3d{
  int size_0;
  int size_1;
  int size_2;
  int length;
  float* data;
  array3d(int size_0,int size_1, int size_2)
    :size_0(size_0),size_1(size_1),size_2(size_2)
  {
    length=size_0*size_1*size_2;
    data=new float[length];
  }
  ~array3d(){
    delete [] data;
  }
  void show(){
    for(int _i0=0; _i0 < size_0; _i0++){
      // printing slice
      for(int _i1=0; _i1 < size_1; _i1++){
        for(int _i2=0; _i2 < size_2; _i2++){
          cout<<data[_i0*size_1*size_2 + _i1*size_2 + _i2]<<" ";
        }cout<<endl;
      }cout<<"------"<<endl;
    }
  }

};

__global__
void add(array2d arr1, array2d arr2)
// void add(int n,float*x, float*y)
{
  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < arr1.width*arr1.height; i+=stride){
    // arr1.data[i] = arr1.data[i]+arr2.data[i];
    // arr1.data[i] = blockIdx.x;

    // unravel index
    int row=i/arr1.width;
    int col=i%arr1.width;

    // arr1.data[row*arr1.width+col]=threadIdx.x;
    arr1.data[row*arr1.width+col]+=arr2.data[row*arr1.width+col];
  }
}
int main(void)
{

  array3d arr(3,5,5);
  for(int i=0; i < arr.length; i++){
    arr.data[i]=0.0;
  }

  int _i0=2;
  int _i1=2;
  int _i2=3;
  arr.data[_i0*arr.size_1*arr.size_2 + _i1*arr.size_2 + _i2]=99;

  int raveled_index=_i0*arr.size_1*arr.size_2 + _i1*arr.size_2 + _i2;
  cout << "raveled_index => " << raveled_index << endl;
  arr.show();
  cout<<" -- "<<endl;
  // unravel index
  int _i_ur_0=raveled_index/(arr.size_1*arr.size_2);
  int _i_ur_1=(raveled_index-(arr.size_1*arr.size_2*_i_ur_0))/(arr.size_2);
  int _i_ur_2=raveled_index%arr.size_2;
  cout << "_i_ur_0 => " << _i_ur_0 << endl;
  cout << "_i_ur_1 => " << _i_ur_1 << endl;
  cout << "_i_ur_2 => " << _i_ur_2 << endl;

  exit(0);
  int N = 10;
  array2d arr1(N,N);
  array2d arr2(N,N);

  // initialize x and y arrays on the host
  for (int i = 0; i < N*N; i++) {
    arr1.data[i] = 3.0f;
    arr2.data[i] = 5.0f;
  }
  // Run kernel on 1M elements on the GPU
  int blockSize=256;
  int numBlocks=(N*N+blockSize-1)/blockSize;
  cout << "numBlocks => " << numBlocks << endl;
  add<<<numBlocks, blockSize>>>(arr1, arr2);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  cout<<"arr1:"<<endl;
  // arr1.print();
  for(int i=0; i < N; i++){
    for(int j=0; j < N; j++){
      cout<<arr1.data[i*arr1.width+j]<<"  ";
    }cout<<endl;
  }
  // cout<<"arr2:"<<endl;
  // arr2.print();
  return 0;
}
