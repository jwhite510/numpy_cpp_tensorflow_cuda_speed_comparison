
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
struct array2d{
  int width;
  int height;
  float* data;
  array2d(int width,int height):width(width),height(height){
    hipMallocManaged(&data,width*height*sizeof(float));
  }
  ~array2d(){
    hipFree(&data);
  }

};
struct array3d{
  int size_0;
  int size_1;
  int size_2;
  int length;
  float* h_data;
  float* d_data;
  array3d(int size_0,int size_1, int size_2)
    :size_0(size_0),size_1(size_1),size_2(size_2)
  {
    length=size_0*size_1*size_2;
    // allocate memory on device
    hipMalloc(&d_data,length*sizeof(float));
    // allocate memory on host
    h_data = new float[length];
    // cudaMallocManaged(&data,length*sizeof(float));
  }
  void CopyToHost(){
    hipMemcpy(h_data,d_data,length*sizeof(float),hipMemcpyDeviceToHost);
  }
  void CopyToDevice(){
    hipMemcpy(d_data,h_data,length*sizeof(float),hipMemcpyHostToDevice);
  }
  ~array3d(){
    delete [] h_data;
    hipFree(d_data);
  }
  void show(){
    for(int _i0=0; _i0 < size_0; _i0++){
      // printing slice
      for(int _i1=0; _i1 < size_1; _i1++){
        for(int _i2=0; _i2 < size_2; _i2++){
          cout<<h_data[_i0*size_1*size_2 + _i1*size_2 + _i2]<<" ";
        }cout<<endl;
      }cout<<"------"<<endl;
    }
  }

};
__device__
float GetElement(const array3d &arr, int i_0,int i_1,int i_2)
{
  return arr.d_data[i_0*arr.size_1*arr.size_2 + i_1*arr.size_2 + i_2];
}
__device__ void SetElement(array3d &arr, int i_0, int i_1, int i_2, float value)
{
  arr.d_data[i_0*arr.size_1*arr.size_2 + i_1*arr.size_2 + i_2]=value;
}

struct Myarray{
  int N;
  float* h_array;
  float* d_array;
};

__global__
void add(Myarray arr)
{
  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < arr.N; i+=stride){
    arr.d_array[i]=2*arr.d_array[i];
  }
}
int main(void)
{

  Myarray myarray;
  myarray.N=100;
  myarray.h_array=new float[myarray.N];
  hipMalloc(&myarray.d_array,myarray.N*sizeof(float));

  int val=0;
  for(int i=0; i < myarray.N; i++){
    myarray.h_array[i]=val++;
  }

  // HOST TO DEVICE
  hipMemcpy(myarray.d_array,myarray.h_array,myarray.N*sizeof(float),hipMemcpyHostToDevice);

  int blockSize=256;
  int numBlocks=(myarray.N+blockSize-1)/blockSize;
  add<<<numBlocks, blockSize>>>(myarray);

  // DEVICE TO HOST
  hipMemcpy(myarray.h_array,myarray.d_array,myarray.N*sizeof(float),hipMemcpyDeviceToHost);

  for(int i=0; i < 10; i++){
    cout<<myarray.h_array[i]<<" ";
  }cout<<endl;

  return 0;
}
