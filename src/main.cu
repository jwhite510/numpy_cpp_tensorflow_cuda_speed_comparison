
#include <hip/hip_runtime.h>
#include <iostream>
#include<chrono>
#include <math.h>

using namespace std;
using namespace std::chrono;

void print10(float* s){
  for(int i=0; i < 10; i++){
    cout<<s[i]<<" ";
  }cout<<endl;
}
// Kernel function to add the elements of two arrays
struct array2d{
  int width;
  int height;
  float* data;
  array2d(int width,int height):width(width),height(height){
    hipMallocManaged(&data,width*height*sizeof(float));
  }

};

__global__
void add(array2d arr1, array2d arr2)
// void add(int n,float*x, float*y)
{
  // int row=blockIdx.y*blockDim.y+threadIdx.y;
  // int col=blockIdx.x*blockDim.x+threadIdx.x;
  // arr1.data[arr1.width*row+col]+=arr2.data[arr2.width*row+col];

  int index=blockIdx.x*blockDim.x+threadIdx.x;
  int stride=blockDim.x*gridDim.x;
  for(int i=index; i < arr1.width; i+=stride){
    arr1.data[i] = arr1.data[i]+arr2.data[i];
  }

}
int main(void)
{
  int N = 10;
  array2d arr1(N,N);

  array2d arr2(N,N);

  // initialize x and y arrays on the host
  for (int i = 0; i < N*N; i++) {
    arr1.data[i] = 3.0f;
    arr2.data[i] = 3.0f;
  }
  // Run kernel on 1M elements on the GPU
  int blockSize=256;
  int numBlocks=(N+blockSize-1)/blockSize;
  cout << "numBlocks => " << numBlocks << endl;
  // dim3 dimBlock(blockSize,blockSize);
  // dim3 dimGrid(arr1.width/dimBlock.x,arr1.height/dimBlock.y);
  // add<<<dimGrid, dimBlock>>>(arr1.data, arr2.data);
  add<<<numBlocks, blockSize>>>(arr1, arr2);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  cout<<"arr1:"<<endl;
  // arr1.print();
  for(int i=0; i < N; i++){
    for(int j=0; j < N; j++){
      cout<<arr1.data[i*arr1.width+j]<<"  ";
    }cout<<endl;
  }
  // cout<<"arr2:"<<endl;
  // arr2.print();
  return 0;
}
